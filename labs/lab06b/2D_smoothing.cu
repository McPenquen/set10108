#include <hip/hip_runtime.h>
#include <>

#include "gpuErrchk.h"

int main(int argc, char **argv)
{
    // Initialise CUDA - select device
    gpuErrchk(hipSetDevice(0));
    
    return 0;
}