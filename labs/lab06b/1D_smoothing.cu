#include "hip/hip_runtime.h"
#include <vector>
#include <iostream>
#include <random>

#include <hip/hip_runtime.h>
#include <>

#include "gpuErrchk.h"

using namespace std;

constexpr size_t AMOUNT = 10;
constexpr size_t THREADS_NUMBER = 5;
constexpr size_t MASK_SIZE = 5;

__global__ void OneDSmooth(const float* input, const float* mask, float* output)
{
    // Get block index
    unsigned int bIdx = blockIdx.x;
    // Get thread index
    unsigned int tIdx = threadIdx.x;
    // Get the number of threads per block
    unsigned int blockSize = blockDim.x;
    // Get the index
    int idx = tIdx + bIdx * blockSize;
    // The radius of the mask
    int maskRadius = ((MASK_SIZE - 1) / 2);
    // The index of the region
    int startingIdx = idx - maskRadius;
    // Size of the region to calculate the smoothing from
    unsigned int maskSize = MASK_SIZE;
    // Overlap of the beginning
    unsigned int startingOverlap = 0;

    __shared__ float maskLocal[5];
    maskLocal[0] = 0.06136f;
    maskLocal[1] = 0.24477f;
    maskLocal[2] = 0.38774f;
    maskLocal[3] = 0.24477f;
    maskLocal[4] = 0.06136f;


    // Adjust the mask size based on the location
    
    if ((idx - maskRadius) < 0) // if it is at the beginning
    {
        while (startingIdx != 0)
        {
            maskSize--;
            startingIdx++;
            startingOverlap++;
        }
    }
    else if ((idx + maskRadius) > (AMOUNT-1))
    {
        unsigned int endingIndx = (idx + maskRadius);
        while (endingIndx != (AMOUNT-1))
        {
            maskSize--;
            endingIndx--;
        }
    }
    
    /*
    if ((idx - maskRadius) == -1)
    {
        startingOverlap = 1;
        maskSize = 4;
        startingIdx = 0;
    }
    else if ((idx - maskRadius) == -2)
    {
        startingOverlap = 2;
        maskSize = 3;
        startingIdx = 0;
    }
    else if ((idx + maskRadius) == (AMOUNT + 1))
    {
        maskSize = 4;
    }
    else if ((idx + maskRadius) == (AMOUNT + 2))
    {
        maskSize = 3;
    }
    */
    // Calculate the result
    float result = 0.0f;
    // The mask has to have 5 elements
    for (int i = 0; i < maskSize; i++)
    {
        result += input[startingIdx + i] * maskLocal[i + startingOverlap];
    }
    result = result / maskSize;
    output[idx] = result;
}

int main(int argc, char **argv)
{
    // Create the host memory
    vector<float> h_Input(AMOUNT);
    vector<float> h_Mask(5);
    vector<float> h_Output(AMOUNT);
    auto dataSize = sizeof(float) * AMOUNT;

    // Random generator
    random_device r;
    default_random_engine e(r());
    uniform_real_distribution<float> distribution(0.0f, 1.0f);

    // Generate random numbers
    for (unsigned int i = 0; i < AMOUNT; i++)
    {
        h_Input[i] = distribution(e);
    }

    // Save the mask
    h_Mask = { 0.06136f, 0.24477f, 0.38774f, 0.24477f, 0.06136f };

    cout << "Input:" << endl;

    //print the input
    for (unsigned int i = 0; i < h_Input.size(); i++)
    {
        cout << i << ": " << h_Input[i] << endl;
    }
    cout << "Mask:" << endl;
    for (unsigned int i = 0; i < MASK_SIZE; i++)
    {
        cout << i << ": " << h_Mask[i] << endl;
    }

    // Buffer
    float* inputBuffer, *outputBuffer, *maskBuffer;

    // Initialise buffers
    hipMalloc((void**)&inputBuffer, dataSize);
    hipMalloc((void**)&maskBuffer, MASK_SIZE);
    hipMalloc((void**)&outputBuffer, dataSize);

    // Write host data to device
    hipMemcpy(inputBuffer, &h_Input[0], dataSize, hipMemcpyHostToDevice);
    hipMemcpy(maskBuffer, &h_Mask[0], MASK_SIZE, hipMemcpyHostToDevice);
    
    // Run the function
    OneDSmooth<<<AMOUNT / THREADS_NUMBER, THREADS_NUMBER>>>(inputBuffer, maskBuffer, outputBuffer);

    // Wait until it's completed
    hipDeviceSynchronize();

    // Read the output buffer
    hipMemcpy(&h_Output[0], outputBuffer, dataSize, hipMemcpyDeviceToHost);

    // Free memory
    hipFree(inputBuffer);
    hipFree(outputBuffer);
    hipFree(maskBuffer);

    cout << "Output:" << endl;

    //print the output
    for (unsigned int i = 0; i < h_Output.size(); i++)
    {
        cout << i << ": " << h_Output[i] << endl;
    }

    float checkup = h_Input[0] * h_Mask[2] + h_Input[1] * h_Mask[3] + h_Input[2] * h_Mask[4];
    checkup /= 3;
    cout << "Checkup for value 0: " << checkup << endl;

    checkup = h_Input[3] * h_Mask[0] + h_Input[4] * h_Mask[1] + h_Input[5] * h_Mask[2] + h_Input[6] * h_Mask[3] + h_Input[7] * h_Mask[4];
    checkup /= 5;

    cout << "Checkup for value 5: " << checkup<< endl;

    return 0;
}